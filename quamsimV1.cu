/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

__global__ void matrix_mul(float *A, float *C, float a, float b, float c, float d, int state_size, int t_bit){
    // A contains input
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    //i=x1, flipped=x2
    int flipped = ((1 << t_bit) | i);
    if (i < state_size){
        if (flipped  > i){
            C[i] = (A[i] * a ) + (A[flipped] * b);
            C[flipped] = (A[i] * c ) + (A[flipped] * d);
        }
    }
}


/**
 * Host main routine
 */

int main(int argc, char **argv){
    /////////////////////////////////////////////////////////////////
    //                    Reading From Input                       //
    /////////////////////////////////////////////////////////////////
    // How many quantum gate matrices we will have? variable or fixed?
    // Six - fixed
    int NUM_QUANTUM_GATES = 6;
    int QUANTUM_GATE_SIZE = 4;
    float gates[NUM_QUANTUM_GATES][QUANTUM_GATE_SIZE];
    int T_BITS[NUM_QUANTUM_GATES];

    FILE* in_file = fopen(argv[1], "r");                   // read only
    // equivalent to saying if ( in_file == NULL )
     if (!in_file){
        printf("oops, file can't be read\n");
        exit(-1);
     }

    // Read and store the quantum gate matrices in a 2-D array called gates
    // Each matrix is reprsented like this:
    // [a,b,c,d] = a  b
    //             c  d
    char input_elem[32];                                   // arbitrary length
    for (int i=0; i < NUM_QUANTUM_GATES; i++){
       for (int j=0; j < QUANTUM_GATE_SIZE; j++){
           int r = fscanf(in_file, "%s", &input_elem[0]);
           if (r == EOF){
               printf("Incorrect input formatting. Abort\n");
               return 1;
           }
           gates[i][j] = atof(input_elem);
       }
    }

    // Read the rest of the file
    int max_vector_size=pow(2,30);
    float* state_vector  = (float*)malloc(max_vector_size * sizeof(float));
    int count = 0;
    while (fscanf(in_file, "%s", &input_elem[0]) != EOF){
        state_vector[count] = atof(input_elem);
        count++;
    }

    // Go to the back of the state_vector, and grabs the
    // last NUM_QUANTUM_GATES elements, convert them into ints
    // and store them in an array
    int tmp;
    for (int i=0; i<NUM_QUANTUM_GATES; i++){
        tmp =  (int)state_vector[count-i-1];
        T_BITS[NUM_QUANTUM_GATES - i - 1] = tmp;
    }

    /////////////////////////////////////////////////////////////////
    //                    CUDA Error Detection                     //
    /////////////////////////////////////////////////////////////////

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = count - NUM_QUANTUM_GATES;
    size_t size = numElements * sizeof(float);
    //printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_C == NULL){
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i){
        h_A[i] = state_vector[i];
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int round=0;
    while (round < 6){
        printf("round %d\n", round);
        float a,b,c,d;
        a=gates[round][0];
        b=gates[round][1];
        c=gates[round][2];
        d=gates[round][3];
        int t_bit = T_BITS[round];

        // Copy the host input vectors A and B in host memory to the device input vectors in
        // device memory
        //printf("Copy input data from the host memory to the CUDA device\n");
        err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }


        // Launch the Vector Add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
        //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        matrix_mul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, a, b, c, d, numElements, t_bit);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Copy the device result vector in device memory to the host result vector
        // in host memory.
        //printf("Copy output data from the CUDA device to the host memory\n");
        err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        memcpy(h_A, h_C, size);
        round++;
    }
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        printf("%.3f\n", h_C[i]);
    }


    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_C);

    //Free IO memory
    free(state_vector);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

